#include "hip/hip_runtime.h"
#pragma unmanaged
#include "orderscan.h"
#include <hip/hip_vector_types.h>
#include <vector>

struct cu_statistic_base
{
  float delta;
  int time;
  int count;
  __device__ cu_statistic_base& operator += (const cu_statistic_base& stat)
  {
    delta += stat.delta;
    time += stat.time;
    count += stat.count;
    return *this;
  }
};

struct cu_statistic
{
  cu_statistic_base profit;
  cu_statistic_base loss;
  cu_statistic_base timeout;
  __device__ cu_statistic& operator += (const cu_statistic& stat)
  {
    this->profit += stat.profit;
    this->loss += stat.loss;
    this->timeout += stat.timeout;
    return *this;
  }
}; 

__device__ float kernelBuy(float2 open, float2 close)
{
  return close.x - open.y;
}

__device__ float kernelSell(float2 open, float2 close)
{
  return open.x - close.y;
}

struct position
{
  float delta;
  int time;
};

enum position_type
{
    BUY, SELL
};

__device__ position kernelSinglePosition(const float2* quotes, int index, int timeout, float takeprofit, float stoploss, position_type ptype)
{
  position pos;
  for (int i = 1; i < timeout; i++)
  {
    float delta = ptype == BUY ? kernelBuy(quotes[index], quotes[index + i]) : kernelSell(quotes[index], quotes[index + i]);
    if (delta >= takeprofit || delta <= -stoploss)
    {
      pos.delta = delta;
      pos.time = i;
      return pos;
    }
  }
  pos.delta = ptype == BUY ? kernelBuy(quotes[index], quotes[index + timeout]) : kernelSell(quotes[index], quotes[index + timeout]);
  pos.time = timeout;
  return pos;
}

__device__ cu_statistic kernelSingleStat(const float2* quotes, int index, int timeout, float takeprofit, float stoploss, char* wstat, position_type ptype)
{
  cu_statistic stat = { {0,0,0}, {0,0,0}, {0,0,0} };
  position pos = kernelSinglePosition(quotes, index, timeout, takeprofit, stoploss, ptype);
  if (pos.time == timeout)
  {
    stat.timeout.delta = pos.delta;
    stat.timeout.time = pos.time;
    stat.timeout.count = 1;
    wstat[index] = 0;
  }
  else if (pos.delta >= takeprofit)
  {
    stat.profit.delta = pos.delta;
    stat.profit.time = pos.time;
    stat.profit.count = 1;
    wstat[index] = 1;
  }
  else if (pos.delta <= -stoploss)
  {
    stat.loss.delta = pos.delta;
    stat.loss.time = pos.time;
    stat.loss.count = 1;
    wstat[index] = 0;
  }
  return stat;
}

__device__ void kernelOrderScan(const float2* quotes, int count, int timeout, float takeprofit, float stoploss, cu_statistic* stat, char* wstat, position_type ptype)
{
  __shared__ cu_statistic block_cache[native::ThreadCount];
  int index = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
  int idx_thread = threadIdx.y * blockDim.x + threadIdx.x;
  if (index < (count - timeout))
  {
    block_cache[idx_thread] = kernelSingleStat(quotes, index, timeout, takeprofit, stoploss, wstat, ptype);
  }
  else
  {
    cu_statistic empty_stat = { {0,0,0}, {0,0,0}, {0,0,0} };
    block_cache[idx_thread] = empty_stat;
  }

  __syncthreads();

  int thread_count = blockDim.x * blockDim.y;
  if (thread_count < native::ThreadCount)
  {
    if (idx_thread == 0)
    {
      for (int i = 1; i < thread_count; i++)
      {
        block_cache[0] += block_cache[i];
      }
    }
    __syncthreads();
  }
  else
  {
    for (int t = native::ThreadCount / 2; t > 0; t /= 2)
    {
      if (idx_thread < t)
      {
        block_cache[idx_thread] += block_cache[idx_thread + t];
      }
      __syncthreads();
    }
  }

  if (idx_thread == 0)
  {
    int idx_block = blockIdx.y * gridDim.x + blockIdx.x;
    stat[idx_block] = block_cache[0];
  }
}

__global__ void kernelOrderBuyScan(const float2* quotes, int count, int timeout, float takeprofit, float stoploss, cu_statistic* stat, char* wstat)
{
  kernelOrderScan(quotes, count, timeout, takeprofit, stoploss, stat, wstat, BUY);
}

__global__ void kernelOrderSellScan(const float2* quotes, int count, int timeout, float takeprofit, float stoploss, cu_statistic* stat, char* wstat)
{
  kernelOrderScan(quotes, count, timeout, takeprofit, stoploss, stat, wstat, SELL);
}

namespace native
{
  CudaOrderScan::CudaOrderScan( quote* quotes, int count, int timeout )
    : Count(count)
    , Timeout(timeout)
    , cuda_dim(count - timeout)
  {
    std::vector<float2> prepared_quotes(count);
    for (int i = 0; i < count; i++)
    {
      prepared_quotes[i].x = quotes[i].low; // bid - ���� �� ������� �� ��������
      prepared_quotes[i].y = quotes[i].high; // ask - ���� �� ������� �� �����
      // TODO: ������� �� ��� ������ spread.
    }
    dev_quotes.reset(CudaCreater<float2>(count), CudaDeleter<float2>);
    hipMemcpy(dev_quotes.get(), prepared_quotes.data(), sizeof(float2) * count, hipMemcpyHostToDevice);
    dev_stat.reset(CudaCreater<cu_statistic>(cuda_dim.Nblocks()), CudaDeleter<cu_statistic>);
    dev_wstat.reset(CudaCreater<char>(count - timeout), CudaDeleter<char>);
  }

  void CudaOrderScan::ScanBuy( float takeprofit, float stoploss )
  {
    kernelOrderBuyScan<<<*cuda_dim.gridSize, *cuda_dim.blockSize>>>(dev_quotes.get(), Count, Timeout, takeprofit, stoploss, dev_stat.get(), dev_wstat.get());
  }

  void CudaOrderScan::ScanSell( float takeprofit, float stoploss )
  {
    kernelOrderSellScan<<<*cuda_dim.gridSize, *cuda_dim.blockSize>>>(dev_quotes.get(), Count, Timeout, takeprofit, stoploss, dev_stat.get(), dev_wstat.get());
  }

  statistic CudaOrderScan::Scan( float takeprofit, float stoploss, ScanFunc func )
  {
    (this->*func)(takeprofit, stoploss);
    std::vector<cu_statistic> partial_stat(cuda_dim.Nblocks());
    hipMemcpy(partial_stat.data(), dev_stat.get(), sizeof(cu_statistic) * cuda_dim.Nblocks(), hipMemcpyDeviceToHost);
    statistic stat;
    for (int i = 0; i < partial_stat.size(); i++)
    {
      stat.Add(partial_stat[i]);
    }
    // ��������� ����� ����.
    std::vector<char> wstat(Count - Timeout);
    hipMemcpy(wstat.data(), dev_wstat.get(), Count - Timeout, hipMemcpyDeviceToHost);
    bool inwindow = false;
    for (int i = 0; i < (Count - Timeout); i++)
    {
      if (wstat[i])
      {
        inwindow = true;
      }
      else
      {
        if (inwindow)
        {
          stat.wcount++;
          inwindow = false;
        }
      }
    }
    if (inwindow)
    {
      stat.wcount++;
      inwindow = false;
    }
    return stat;
  }

  void statistic::Add( const cu_statistic& stat )
  {
    this->profit.Add(stat.profit);
    this->loss.Add(stat.loss);
    this->timeout.Add(stat.timeout);
  }

  void statistic::statistic_base::Add( const cu_statistic_base& stat )
  {
    avg_delta += stat.delta;
    avg_wait += stat.time;
    count += stat.count;
  }

}