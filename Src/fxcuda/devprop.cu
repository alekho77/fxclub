#pragma unmanaged
#include "devprop.h"

namespace native
{
    int GetDeviceCount()
    {
        int count;
        if (hipGetDeviceCount(&count) == hipSuccess)
        {
            return count;
        }
        return -1;
    }

    int CurrentDevice()
    {
        int dev;
        if (hipGetDevice(&dev) == hipSuccess)
        {
            return dev;
        }
        return -1;
    }

    std::tr1::shared_ptr<DeviceProp> GetDeviceProp( int dev )
    {
        std::tr1::shared_ptr<DeviceProp> dev_prop;
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, dev) == hipSuccess)
        {
            dev_prop.reset(new DeviceProp);
            dev_prop->Name = prop.name;
            dev_prop->GlobalMem = prop.totalGlobalMem;
            dev_prop->SharedMem = prop.sharedMemPerBlock;
            dev_prop->ConstMem = prop.totalConstMem;
            dev_prop->ThreadsPerBlock = prop.maxThreadsPerBlock;
            memcpy(dev_prop->ThreadsDim, prop.maxThreadsDim, sizeof(prop.maxThreadsDim));
            memcpy(dev_prop->GridSize, prop.maxGridSize, sizeof(prop.maxGridSize));
            dev_prop->ClockRate = prop.clockRate;
            dev_prop->Major = prop.major;
            dev_prop->Minor = prop.minor;
            dev_prop->ProcessorCount = prop.multiProcessorCount;
            dev_prop->CurrentKernels = prop.concurrentKernels;
            dev_prop->MemoryClockRate = prop.memoryClockRate;
            dev_prop->ThreadsPerProcessor = prop.maxThreadsPerMultiProcessor;
        }
        return dev_prop;
    }
}
