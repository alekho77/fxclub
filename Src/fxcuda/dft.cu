#include "hip/hip_runtime.h"
#pragma unmanaged
#include "dft.h"

__global__ void kernelCreateTurner(float* turners, int n)
{
  int idx = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
  if (idx < n)
  {
    float a = 6.283185307179586476925286766559f * __int2float_rn(idx) / __int2float_rn(n);
    float* cs = turners + 2 * idx;
    sincosf(a, cs + 1, cs);
  }
}

__global__ void kernelHarmonic(float* x, float* coef, int k, int n, float* turners)
{
  __shared__ float2 block_cache[native::ThreadCount];
  int idx = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
  int idx_thread = threadIdx.y * blockDim.x + threadIdx.x;
  if (idx < n)
  {
    int idx_turner = 2 * ( ( (long long)(idx) * (long long)(k) ) % (long long)(n) );
    block_cache[idx_thread] = make_float2(x[idx] * turners[idx_turner], - x[idx] * turners[idx_turner + 1]);
  }
  else
  {
    block_cache[idx_thread] = make_float2(0, 0);
  }

  __syncthreads();

  int thread_count = blockDim.x * blockDim.y;
  if (thread_count < native::ThreadCount)
  {
    if (idx_thread == 0)
    {
      for (int i = 1; i < thread_count; i++)
      {
        block_cache[0].x += block_cache[i].x;
        block_cache[0].y += block_cache[i].y;
      }
    }
    __syncthreads();
  }
  else
  {
    for (int t = native::ThreadCount / 2; t > 0; t /= 2)
    {
      if (idx_thread < t)
      {
        block_cache[idx_thread].x += block_cache[idx_thread + t].x;
        block_cache[idx_thread].y += block_cache[idx_thread + t].y;
      }
      __syncthreads();
    }
  }

  if (idx_thread == 0)
  {
    int idx_block = blockIdx.y * gridDim.x + blockIdx.x;
    coef[2 * idx_block + 0] = block_cache[0].x;
    coef[2 * idx_block + 1] = block_cache[0].y;
  }
}

namespace native
{
  CudaDFourier::CudaDFourier( float* x, int n ) : N(n), cuda_dim(n)
  {
    devX.reset(CudaCreater<float>(n), CudaDeleter<float>);
    hipMemcpy(devX.get(), x, sizeof(float) * n, hipMemcpyHostToDevice);
    devTurners.reset(CudaCreater<float>(2*n), CudaDeleter<float>);
    kernelCreateTurner<<<*cuda_dim.gridSize, *cuda_dim.blockSize>>>(devTurners.get(), N);
  }

  Complex CudaDFourier::Harmonic( int k )
  {
    Complex coef(0, 0);
    const int Nblocks = cuda_dim.gridSize->x * cuda_dim.gridSize->y;
    std::tr1::shared_ptr<float> dev_coefs(CudaCreater<float>(2 * Nblocks), CudaDeleter<float>);
    kernelHarmonic<<<*cuda_dim.gridSize, *cuda_dim.blockSize>>>(devX.get(), dev_coefs.get(), k, N, devTurners.get());
    
    std::vector<float> coefs(2 * Nblocks);
    hipMemcpy(&coefs[0], dev_coefs.get(), sizeof(float) * (2 * Nblocks), hipMemcpyDeviceToHost);
    for (int i = 0; i < Nblocks; i++)
    {
      coef += Complex(coefs[2 * i], coefs[2 * i + 1]);
    }
    return coef / (float)N;
  }

}